#include "hip/hip_runtime.h"
#include "code_objects/neurongroup_10_stateupdater_codeobject_5.h"
#include "objects.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>



////// SUPPORT CODE ///////
namespace {
    randomNumber_t _host_rand(const int _vectorisation_idx);
    randomNumber_t _host_randn(const int _vectorisation_idx);
    int32_t _host_poisson(double _lambda, const int _vectorisation_idx);

    ///// block extra_device_helper /////

    ///// support_code_lines /////
        
    template <typename T>
    __host__ __device__
    double _brian_exp(T value)
    {
    #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
        return exp((double)value);
    #else
        return exp(value);
    #endif
    }
    inline __host__ __device__
    float _brian_exp(float value)
    {
        return exp(value);
    }
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                    inline __device__ int _brian_atomicAdd(int* address, int val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ float _brian_atomicAdd(float* address, float val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ double _brian_atomicAdd(double* address, double val)
                    {
                            #if (__CUDA_ARCH__ >= 600)
            // hardware implementation
            return atomicAdd(address, val);
                            #else
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val +
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                            #endif
                    }
                    inline __device__ int _brian_atomicMul(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val * assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicMul(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val *
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicMul(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val *
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }
                    inline __device__ int _brian_atomicDiv(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val / assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicDiv(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val /
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicDiv(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val /
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }


    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    randomNumber_t _host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    randomNumber_t _host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    int32_t _host_poisson(double _lambda, const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_poisson` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
}

////// hashdefine_lines ///////



__global__ void
_run_kernel_neurongroup_10_stateupdater_codeobject_5(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_neurongroup_10_A,
	double* _ptr_array_neurongroup_10_Iexc,
	double* _ptr_array_neurongroup_10_Iinh,
	double* _ptr_array_neurongroup_10_X,
	double* _ptr_array_neurongroup_10_Y,
	const double _value_array_defaultclock_dt,
	double* _ptr_array_neurongroup_10_v
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * blockDim.x + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _numA = 16;
	const double Cm = 2.81e-10;
	const double DeltaT = 0.002;
	const double EL = - 0.0706;
	const int _numIexc = 16;
	const int _numIinh = 16;
	const int64_t N = 16;
	const double Vth = - 0.0504;
	const int _numX = 16;
	const int _numY = 16;
	const double c = 4e-09;
	const double gL = 3.0000000000000004e-08;
	const double tau_A = 0.001;
	const double tau_decay = 0.05;
	const double tau_rise = 0.005;
	const int _numv = 16;

    ///// kernel_lines /////
        
    const double* _ptr_array_defaultclock_dt = &_value_array_defaultclock_dt;



    if(_vectorisation_idx >= _N)
    {
        return;
    }



    ///// scalar_code /////
        
    const double dt = _ptr_array_defaultclock_dt[0];
    const double _lio_1 = 1.0f*dt/tau_A;
    const double _lio_2 = - EL;
    const double _lio_3 = 1.0f*1.0/tau_decay;
    const double _lio_4 = 1.0f*1.0/tau_rise;
    const double _lio_5 = 1.0f*dt/tau_decay;
    const double _lio_6 = 1.0f*dt/Cm;
    const double _lio_7 = DeltaT * gL;
    const double _lio_8 = 1.0f*1.0/DeltaT;
    const double _lio_9 = - Vth;


    {
        ///// vector_code /////
                
        double A = _ptr_array_neurongroup_10_A[_idx];
        const double Iexc = _ptr_array_neurongroup_10_Iexc[_idx];
        const double Iinh = _ptr_array_neurongroup_10_Iinh[_idx];
        double X = _ptr_array_neurongroup_10_X[_idx];
        double Y = _ptr_array_neurongroup_10_Y[_idx];
        const double dt = _ptr_array_defaultclock_dt[0];
        double v = _ptr_array_neurongroup_10_v[_idx];
        const double _A = A + (_lio_1 * ((- A) + (c * (_lio_2 + v))));
        const double _X = X + (dt * ((_lio_3 * (- X)) + (_lio_4 * Y)));
        const double _Y = (_lio_5 * (- Y)) + Y;
        const double _v = v + (_lio_6 * (((((- A) + (_lio_7 * _brian_exp(_lio_8 * (_lio_9 + v)))) + Iexc) + Iinh) - (gL * (_lio_2 + v))));
        A = _A;
        X = _X;
        Y = _Y;
        v = _v;
        _ptr_array_neurongroup_10_A[_idx] = A;
        _ptr_array_neurongroup_10_X[_idx] = X;
        _ptr_array_neurongroup_10_Y[_idx] = Y;
        _ptr_array_neurongroup_10_v[_idx] = v;


    }
}



void _run_neurongroup_10_stateupdater_codeobject_5()
{
    using namespace brian;


    ///// HOST_CONSTANTS ///////////
    const int _numA = 16;
		const double Cm = 2.81e-10;
		const double DeltaT = 0.002;
		const double EL = - 0.0706;
		const int _numIexc = 16;
		const int _numIinh = 16;
		const int64_t N = 16;
		const double Vth = - 0.0504;
		const int _numX = 16;
		const int _numY = 16;
		const double c = 4e-09;
		const double gL = 3.0000000000000004e-08;
		const double tau_A = 0.001;
		const double tau_decay = 0.05;
		const double tau_rise = 0.005;
		const int _numv = 16;

    const int _N = N;

    ///// ADDITIONAL_HOST_CODE /////
    


    static int num_threads, num_blocks;
    static size_t needed_shared_memory = 0;
    static bool first_run = true;
    if (first_run)
    {
        // get number of blocks and threads
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    _run_kernel_neurongroup_10_stateupdater_codeobject_5, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;





        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    _run_kernel_neurongroup_10_stateupdater_codeobject_5, num_threads, needed_shared_memory)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(_run_kernel_neurongroup_10_stateupdater_codeobject_5))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "_run_kernel_neurongroup_10_stateupdater_codeobject_5 "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        _run_kernel_neurongroup_10_stateupdater_codeobject_5, num_threads, needed_shared_memory)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }


        else
        {
            printf("INFO _run_kernel_neurongroup_10_stateupdater_codeobject_5\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per thread\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks,
                   num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }


    _run_kernel_neurongroup_10_stateupdater_codeobject_5<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            dev_array_neurongroup_10_A,
			dev_array_neurongroup_10_Iexc,
			dev_array_neurongroup_10_Iinh,
			dev_array_neurongroup_10_X,
			dev_array_neurongroup_10_Y,
			_array_defaultclock_dt[0],
			dev_array_neurongroup_10_v
        );

    CUDA_CHECK_ERROR("_run_kernel_neurongroup_10_stateupdater_codeobject_5");


}


